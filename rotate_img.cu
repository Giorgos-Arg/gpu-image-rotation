
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define CHECK(e) { int res = (e); if (res) printf("CUDA ERROR %d\n", res); }
#define CHANNEL 3

// Struct for measuring performance
struct GpuTimer{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer(){
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer(){
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start(){
		hipEventRecord(start, 0);
	}

	void Stop(){
		hipEventRecord(stop, 0);
	}

	float Elapsed(){
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

struct Image {
	int width;
	int height;
	unsigned int bytes;
	unsigned char *img;
	unsigned char *dev_img;
};

// Reads a color ppm image file and saves the data in the provided Image structure. 
// The max_col_val is set to the value read from the input file.
// This is used later for writing the output image. 
int readInpImg(const char * fname, Image & source, int & max_col_val) {
	FILE *src;
	if (!(src = fopen(fname, "rb"))){
		printf("Couldn't open file %s for reading.\n", fname);
		return 1;
	}
	char p, s;
	fscanf(src, "%c%c\n", &p, &s);
	if (p != 'P' || s != '6'){	// Is it a valid format?
		printf("Not a valid PPM file (%c %c)\n", p, s);
		exit(1);
	}
	fscanf(src, "%d %d\n", &source.width, &source.height);
	fscanf(src, "%d\n", &max_col_val);
	int pixels = source.width * source.height;
	source.bytes = pixels * CHANNEL;  // CHANNEL = 3 => colored image with r, g, and b channels 
	source.img = (unsigned char *)malloc(source.bytes);
	if (fread(source.img, sizeof(unsigned char), source.bytes, src) != source.bytes){
		printf("Error reading file.\n");
		exit(1);
	}
	fclose(src);
	return 0;
}

// Write a color ppm image into a file.  
// Image structure represents the image in the memory. 
int writeOutImg(const char * fname, const Image & rotated, const int max_col_val) {
	FILE *out;
	if (!(out = fopen(fname, "wb"))){
		printf("Couldn't open file for output.\n");
		return 1;
	}
	fprintf(out, "P6\n%d %d\n%d\n", rotated.width, rotated.height, max_col_val);
	if (fwrite(rotated.dev_img, sizeof(unsigned char), rotated.bytes, out) != rotated.bytes){
		printf("Error writing file.\n");
		return 1;
	}
	fclose(out);
	return 0;
}

// To be launched on CPU
void rotate_90_CPU(unsigned char in[], unsigned char out[], int height, int width) {
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			int index_in = i * width * CHANNEL + j * CHANNEL;
			int index_out = j * height * CHANNEL + height * CHANNEL - (i + 1) * CHANNEL;
			out[index_out] = in[index_in];
			out[index_out + 1] = in[index_in + 1];
			out[index_out + 2] = in[index_in + 2];
		}
	}
}

// To be launched on a single thread
__global__ void rotate_90_serial(unsigned char in[], unsigned char out[], int height, int width) {
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			int index_in = i * width * CHANNEL + j * CHANNEL;
			int index_out = j * height * CHANNEL + height * CHANNEL - (i + 1) * CHANNEL;
			out[index_out] = in[index_in];
			out[index_out + 1] = in[index_in + 1];
			out[index_out + 2] = in[index_in + 2];
		}
	}
}

// To be launched with one thread per element
__global__ void rotate_90_parallel_per_element(unsigned char in[], unsigned char out[], int height, int width) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int index_in = i * width * CHANNEL + j * CHANNEL;
	int index_out = j * height * CHANNEL + height * CHANNEL - (i + 1) * CHANNEL;
	int image_size = height * width * CHANNEL;
	if(index_in < image_size && index_in >= 0 && index_out < image_size && index_out >= 0){
		out[index_out] = in[index_in];
		out[index_out + 1] = in[index_in + 1];
		out[index_out + 2] = in[index_in + 2];
	}
}

// To be launched with one thread per element. Thread blocks read & write tiles in coalesced fashion.
__global__ void rotate_90_parallel_per_element_tiled(unsigned char in[], unsigned char out[], int height, int width, unsigned int tile_size) {
	// (i,j) locations of the tile corners for input & output matrices:
	int in_corner_i = blockIdx.x * tile_size, in_corner_j = blockIdx.y * tile_size * CHANNEL;
	int out_corner_i = blockIdx.y * tile_size * CHANNEL, out_corner_j = blockIdx.x * tile_size;
	int x = threadIdx.x, y = threadIdx.y;
	int index_tile = x * (tile_size)*CHANNEL + y * CHANNEL;
	int index_in = in_corner_i * width * CHANNEL + x * width *CHANNEL + in_corner_j + y * CHANNEL;
	int image_size = height * width * CHANNEL;
	int total_tile_size = tile_size * tile_size * CHANNEL;
	extern __shared__ unsigned char tile[];
	// Coalesced read from global mem, rotated write into shared mem:
	if(index_tile < total_tile_size && index_tile >= 0 && index_in < image_size  && index_in >= 0){
		tile[index_tile] = in[index_in];
		tile[index_tile+1] = in[index_in+1];
		tile[index_tile+2] = in[index_in+2];
	}

	__syncthreads();
	int index_out = out_corner_i*height + y * height * CHANNEL + height * CHANNEL - (x+1) * CHANNEL - out_corner_j * CHANNEL;
	// Read from shared mem, coalesced write to global mem:
	if(index_tile < total_tile_size && index_tile >= 0 && index_out < image_size && index_out >= 0 && index_in < image_size  && index_in >= 0){
		out[index_out] = tile[index_tile];
		out[index_out + 1] = tile[index_tile + 1];
		out[index_out + 2] = tile[index_tile + 2];
	}

}

// To be launched with one thread per element. Thread blocks read & write tiles in coalesced fashion.
// Shared memory array padded to avoid bank conflicts.
__global__ void rotate_90_parallel_per_element_tiled_padded(unsigned char in[], unsigned char out[], int height, int width, unsigned int tile_size) {
	// (i,j) locations of the tile corners for input & output matrices:
	int in_corner_i = blockIdx.x * tile_size, in_corner_j = blockIdx.y * tile_size * CHANNEL;
	int out_corner_i = blockIdx.y * tile_size * CHANNEL, out_corner_j = blockIdx.x * tile_size;
	int x = threadIdx.x, y = threadIdx.y;
	int image_size = height * width * CHANNEL;
	int total_tile_size = tile_size * (tile_size + 1) * CHANNEL ;
	int index_tile = x * (tile_size + 1)*CHANNEL + y * CHANNEL;
	int index_in = in_corner_i * width * CHANNEL + x * width *CHANNEL + in_corner_j + y * CHANNEL;
	extern __shared__ unsigned char tile[];
	// coalesced read from global mem, rotated write into shared mem:
	if(index_tile < total_tile_size && index_tile >= 0 && index_in < image_size){
		tile[index_tile] = in[index_in];
		tile[index_tile+1] = in[index_in+1];
		tile[index_tile+2] = in[index_in+2];
	}
	__syncthreads();
	int index_out = out_corner_i*height + y * height * CHANNEL + height * CHANNEL - (x+1)*CHANNEL - out_corner_j*CHANNEL;
	// Read from shared mem, coalesced write to global mem:
	if(index_tile < total_tile_size && index_tile >= 0 && index_out < image_size && index_out >= 0 && index_in < image_size && index_in >=0 ){
		out[index_out] = tile[index_tile];
		out[index_out + 1] = tile[index_tile + 1];
		out[index_out + 2] = tile[index_tile + 2];
	}
}


int main(int argc, char **argv){
	if (argc != 3){
		printf("Usage: exec filename kernel\n");
		exit(1);
	}
	char *fname = argv[1];
	char kname[100] = ""; // kernel name
	int choice = atoi(argv[2]); // kernel choice
	Image source;
	int max_col_val;
	GpuTimer timer;
	unsigned char *d_in, *d_out;
	// Read the input file
	if (readInpImg(fname, source, max_col_val) != 0)  
		exit(1);
	source.dev_img = (unsigned char *)malloc(source.bytes);
	CHECK(hipMalloc(&d_in, source.bytes));
	CHECK(hipMalloc(&d_out, source.bytes));
	CHECK(hipMemcpy(d_in, source.img, source.bytes, hipMemcpyHostToDevice));
	// Run selected kernel
	switch (choice) {
	case 1: // Serial execution on GPU, i.e. creating ONLY ONE thread
		strcpy(kname, "Serial execution on GPU");
		timer.Start();
		rotate_90_serial <<<1, 1>>> (d_in, d_out, source.height, source.width);
		timer.Stop();
		CHECK(hipMemcpy(source.dev_img, d_out, source.bytes, hipMemcpyDeviceToHost));
		break;
	case 2: { // One thread per pixel
		strcpy(kname, "One thread per pixel");
		int k = 32;
		dim3 blocks(ceil((float)source.height / (float)k), ceil((float)source.width / (float)k));   // blocks per grid (using ceil in case height or width are not multiple of k)
		dim3 threads(k, k);	// threads per block
		timer.Start();
		rotate_90_parallel_per_element <<<blocks, threads>>> (d_in, d_out, source.height, source.width);
		timer.Stop();
		CHECK(hipMemcpy(source.dev_img, d_out, source.bytes, hipMemcpyDeviceToHost));
		break;
	}
	case 3: { // One thread per pixel - tiled (16 X 16)
		strcpy(kname, "One thread per pixel - tiled (16 X 16)");
		int k = 16; // tile size is k x k
		unsigned int shmem_size = k * k * CHANNEL * sizeof(unsigned char);
		dim3 blocks(ceil((float)source.height / (float)k), ceil((float)source.width / (float)k));   // blocks per grid 
		dim3 threads(k, k);	// threads per block
		timer.Start();
		rotate_90_parallel_per_element_tiled <<<blocks, threads, shmem_size>>> (d_in, d_out, source.height, source.width, k);
		timer.Stop();
		CHECK(hipMemcpy(source.dev_img, d_out, source.bytes, hipMemcpyDeviceToHost));
		break;
	}
	case 4: { // One thread per matrix element - tiled (16x16) - no shared mem conflict
		strcpy(kname, "One thread per matrix element - tiled (16x16) - no shared mem conflict");
		int k = 16; // tile size is k x k
		dim3 blocks(ceil((float)source.height / (float)k), ceil((float)source.width / (float)k));   // blocks per grid
		dim3 threads(k, k);	// threads per block
		unsigned int shmem_size = k * (k+1) * CHANNEL * sizeof(unsigned char);
		timer.Start();
		rotate_90_parallel_per_element_tiled_padded <<<blocks, threads, shmem_size>>> (d_in, d_out, source.height, source.width, k);
		timer.Stop();
		CHECK(hipMemcpy(source.dev_img, d_out, source.bytes, hipMemcpyDeviceToHost));
		break;
	}
	default:
		printf("Choose a kernel between 1 and 4");
		exit(1);
	}
	printf("\nRotating Image \"%s\" with Height = %d and Width = %d.\nUsing kernel %d: %s\nElapsed time: %g ms.\n\n", argv[1],
		source.height, source.width, choice, kname, timer.Elapsed());
	// Swap height and width for the rotated image
	int temp = source.height;
	source.height = source.width;
	source.width = temp;
	// Write the output file
	if (writeOutImg("rotated.ppm", source, max_col_val) != 0) // For demonstration, the input file is written to a new file named "rotated.ppm" 
		exit(1);
	// free up the allocated memory
	free(source.img);
	free(source.dev_img);
	CHECK(hipFree(d_in));
	CHECK(hipFree(d_out));
	exit(0);
}